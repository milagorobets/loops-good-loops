#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "utils.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    checkCudaErrors(hipSetDevice(0));

    // Allocate GPU buffers for three vectors (two input, one output)    .
	checkCudaErrors(hipMalloc((void**)&dev_c, size * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_a, size * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_b, size * sizeof(int)));

    // Copy input vectors from host memory to GPU buffers.
    checkCudaErrors(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    checkCudaErrors(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

	Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//#ifndef __HIPCC__
//#define __HIPCC__
//#endif

#include <stdio.h>
#include "cPFkernel.cuh"
#include "utils.h"
#include "common.h"

//__constant__ double src_amplitude = 1.0;
//__constant__ double src_frequency = 1.0;

__global__ void PF_iteration_kernel(hipPitchedPtr mPtr, hipExtent mExt, dim3 matrix_dimensions, 
									double src, dim3 srcloc, bool * wallLoc, float * WWall, float * W,
									hipPitchedPtr nmPtr)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//// Find location within the pitched memory
	char *m = (char*)mPtr.ptr;
	char *nm = (char*)nmPtr.ptr;
	int sof = sizeof(float);
	size_t pitch = mPtr.pitch;
	size_t slice_pitch = pitch*mExt.height;
	///*float src = src_amplitude * sin(2 * PI * src_frequency * (double)(t) * 0.01);*/
	char* m_addroff = m + y * pitch + x * sof;
//	char* m1_addroff = m_addroff + 1 * slice_pitch; // Run kernel during init to set these up?
	char* nm_addroff = nm + y * pitch + x * sof;
	//char* m_ptr = m_addroff + 0 * slice_pitch;
	//char * current_slice = m + 1 * slice_pitch; // z = slice number; current_slice = address of beginning of slice
	//	for (int y = 0; y < matrix_dimensions.y; ++y)
	//	{
	//float * current_row = (float*)(current_slice + y * pitch); // address of the beginning of row in a slice
	//		for (int x = 0; x < matrix_dimensions.x; ++x)
	//		{
	//			printf("Current element at %d, %d, %d: %f \n", x, y, z, current_row[x]);
	//		}
	//printf("current elem uo at %d %d is %f \n", x, y, current_row[x]);
	//printf("current elem ao at %d %d is %f \n", x, y, *m1_addroff);
	char * m_ptr = m_addroff;
	float m0 = *(float*)(m_ptr);
	m_ptr += slice_pitch; // just inc m_ptr by slice_pitch
	float m1 = *(float*)(m_ptr);
	m_ptr += slice_pitch;
	float m2 = *(float*)(m_ptr);
	m_ptr += slice_pitch;
	float m3 = *(float*)(m_ptr);

	float newF[4];
	size_t row_w = pitch;

	// Check if source
	if ((x == srcloc.x) && (y = srcloc.y))
	{
		m0 = src; m1 = src; m2 = src; m3 = src;
	}

	// Check if wall
	bool isWall = wallLoc[x + y*matrix_dimensions.x];
	if (isWall)
	{
		// prefetch WWall into __shared__
		newF[0] = WWall[0]*m0 + WWall[1]*m1 + WWall[2]*m2 + WWall[3]*m3;
		newF[1] = WWall[4]*m0 + WWall[5]*m1 + WWall[6]*m2 + WWall[7]*m3;
		newF[2] = WWall[8]*m0 + WWall[9]*m1 + WWall[10]*m2 + WWall[11]*m3;
		newF[3] = WWall[12]*m0 + WWall[13]*m1 + WWall[14]*m2 + WWall[15]*m3;
	}
	else
	{
		// prefetch W into __shared__
		newF[0] = W[0]*m0 + W[1]*m1 + W[2]*m2 + W[3]*m3;
		newF[1] = W[4]*m0 + W[5]*m1 + W[6]*m2 + W[7]*m3;
		newF[2] = W[8]*m0 + W[9]*m1 + W[10]*m2 + W[11]*m3;
		newF[3] = W[12]*m0 + W[13]*m1 + W[14]*m2 + W[15]*m3;
	}

	if (x < MATRIX_DIM-1) *(float*)(nm_addroff + sof) = newF[1];					// if (x < MATRIX_DIM-1) nm0[x+1][y] = newF[1];
	if (x > 0) *(float*)(nm_addroff - sof + slice_pitch) = newF[0];					// if (x > 0) nm1[x-1][y] = newF[0];
	if (y < MATRIX_DIM-1) *(float*)(nm_addroff + pitch + 2 * slice_pitch) = newF[3];		// if (y < MATRIX_DIM-1) nm2[x][y+1] = newF[3];
	if (y > 0) *(float*)(nm_addroff - pitch + 3*slice_pitch) = newF[2];						// if (y > 0) nm3[x][y-1] = newF[2];		

	// sync after calculating nms
	__syncthreads();

	// Copy nm to m
	*(m_addroff) = *(nm_addroff);										// m0[x][y] = nm0[x][y]
	*(m_addroff + 1 * slice_pitch) = *(nm_addroff + 1 * slice_pitch);	// m1[x][y] = nm1[x][y]
	*(m_addroff + 2 * slice_pitch) = *(nm_addroff + 2 * slice_pitch);	// m2[x][y] = nm2[x][y]
	*(m_addroff + 3 * slice_pitch) = *(nm_addroff + 3 * slice_pitch);	// m3[x][y] = nm3[x][y]

	// Edge Cases
	if (x == 0)
	 {
		if (*(nm_addroff) == 0) // if (nm0[0][y] == 0)
		{
			*(m_addroff) = *(nm_addroff + sof); // m0[0][y] = nm0[1][y];
		}
	 }
	 if (x == MATRIX_DIM-1)
	 {
		if (*(nm_addroff + slice_pitch) == 0) // if (nm1[MATRIX_DIM-1][y] == 0)
		{	
			*(m_addroff + slice_pitch) = *(nm_addroff + slice_pitch - sof); // m1[MATRIX_DIM-1][y] = nm1[MATRIX_DIM-2][y];
		}
	 }
	 if (y == 0)
	 {
		if (*(nm_addroff + 2 * slice_pitch) == 0) // if (nm2[x][0] == 0)
		{
			*(m_addroff + 2 * slice_pitch) = *(nm_addroff + 2 * slice_pitch + pitch); // m2[x][0] = nm2[x][1];
		}
	 }
	 if (y == MATRIX_DIM-1)
	 {
		 if (*(nm_addroff + 3 * slice_pitch) == 0) // (nm3[x][MATRIX_DIM-1] == 0)
		{
			*(m_addroff + 3 * slice_pitch) = *(nm_addroff + 3 * slice_pitch - pitch);	// m3[x][MATRIX_DIM-1] = nm3[x][MATRIX_DIM-2];
		}
	 }

	printf("Location %d, %d, m0 = %f \n", x, y, *m_addroff);
	 
	/* m0[x][y] = nm0[x][y];
			m1[x][y] = nm1[x][y];
			m2[x][y] = nm2[x][y];
			m3[x][y] = nm3[x][y];

			if (nm0[0][y] == 0)
			{
				m0[0][y] = nm0[1][y];
			}
			if (nm2[x][0] == 0)
			{
				m2[x][0] = nm2[x][1];
			}
			if (nm1[MATRIX_DIM-1][y] == 0)
			{
				m1[MATRIX_DIM-1][y] = nm1[MATRIX_DIM-2][y];
			}
			if (nm3[x][MATRIX_DIM-1] == 0)
			{
				m3[x][MATRIX_DIM-1] = nm3[x][MATRIX_DIM-2];
			}*/

	//printf("Current m0 element at %d, %d is %f \n", x, y, m0[x]);
	
	//char *m = (char*)mPtr.ptr;
	//size_t pitch = mPtr.pitch; // x padded to power of 2
	////size_t pitch = 4*128;
	////size_t slice_pitch = pitch*mExt.height;
	//size_t slice_pitch = pitch * matrix_dimensions.y; // padded_x*y, whole slice

	//for (int z = 0; z < matrix_dimensions.z; ++z)
	//{
	//	char * current_slice = m + z * slice_pitch; // z = slice number; current_slice = address of beginning of slice
	//	for (int y = 0; y < matrix_dimensions.y; ++y)
	//	{
	//		float * current_row = (float*)(current_slice + y * pitch); // address of the beginning of row in a slice
	//		for (int x = 0; x < matrix_dimensions.x; ++x)
	//		{
	//			printf("Current element at %d, %d, %d: %f \n", x, y, z, current_row[x]);
	//		}
	//	}
	//}

	 /*for(int k = 0; k < mExt.depth; k++)
	{
		char* slice = m + k*slice_pitch;
		for (int j = 0; j < mExt.height; j++)
		{
			for (int 
			float* row = (float*) (slice+j*pitch);
			printf("%f \n", *row);
		}
	}*/

}

#define WWAL_DIMx 4
#define WWAL_DIMy WWAL_DIMx
#define W_DIMx 4
#define W_DIMy W_DIMx

#define BLOCK_DIMx ((MATRIX_DIM>32)?32:MATRIX_DIM)
#define BLOCK_DIMy  BLOCK_DIMx
#define GRID_DIMx ((MATRIX_DIM + BLOCK_DIMx - 1)/BLOCK_DIMx)
#define GRID_DIMy ((MATRIX_DIM + BLOCK_DIMy - 1)/BLOCK_DIMy)

bool * host_Wall;
float * host_WWall;
float * host_W;

double coef = 1.0;

int gpu_iterations;

float *m_host;

double src_amplitude;
double src_frequency;
dim3 src_loc;

void cPFcaller(unsigned int num_iterations, float * m_ptr)
{
	gpu_iterations = num_iterations;
	hipError_t status = hipSuccess;
	dim3 matdim;
	matdim.x = MATRIX_DIM;
	matdim.y = MATRIX_DIM;
	matdim.z = 4;

	dim3 threads(BLOCK_DIMx,BLOCK_DIMy,1);
	dim3 grids(GRID_DIMx,GRID_DIMy,1);

	// Allocate 3D array for m0-m3 (all together)
	hipExtent m_extent = make_hipExtent(sizeof(float)*matdim.x, matdim.y, matdim.z); // width, height, depth
	hipPitchedPtr m_device;
	hipMalloc3D(&m_device, m_extent);
	hipMemcpy3DParms m_p = {0};
	m_host = (float *)malloc(sizeof(float)*MATRIX_DIM*MATRIX_DIM*4); // need to initialize this somehow
	memset(m_host, 0, sizeof(float)*MATRIX_DIM*MATRIX_DIM*4); // set all to 0  -- do it on gpu cudamemset
	m_ptr = m_host; // So that the class can access M values

	// assign some values to all array elements (testing):
	 for (int z = 0; z < 4; z++)
	{
		for (int y = 0; y < MATRIX_DIM; y++)
		{
			for (int x = 0; x < MATRIX_DIM; x++)
			{
				//m_host[z*MATRIX_DIM*MATRIX_DIM+y*MATRIX_DIM+x] = x+y*MATRIX_DIM+z*MATRIX_DIM*MATRIX_DIM;
				m_host[z*MATRIX_DIM*MATRIX_DIM+y*MATRIX_DIM+x] = 0;
				//printf("m_host at %d,%d,%d is %f \n", x,y,z, m_host[z*MATRIX_DIM*MATRIX_DIM+y*MATRIX_DIM+x]);
			}
		}
	}
	
	m_p.srcPtr = make_hipPitchedPtr((void*)m_host, matdim.x*sizeof(float), matdim.x, matdim.y); // ptr, pitch (x*size), x, y
	m_p.dstPtr = m_device;
	m_p.extent = m_extent; // x, y, z
	m_p.kind = hipMemcpyHostToDevice;
	status = hipMemcpy3D(&m_p); 
	if (status != hipSuccess){printf("M MemcpyHtD: %s \n", hipGetErrorString(status));}
	hipMemset3D(m_device, 0, m_extent);

	// Allocate 3D array for nm0-nm3
	hipExtent nm_extent = make_hipExtent(sizeof(float)*matdim.x, matdim.y, matdim.z);
	hipPitchedPtr nm_device;
	hipMalloc3D(&nm_device, nm_extent); // don't need to init to 0 'cause we will just overwrite it anyways
	hipMemset3D(nm_device, 0, nm_extent);
	
	// Allocate 2D array for wall (unrolled to 1D)
	bool * dev_wall;
	status = hipMalloc((void**)&dev_wall, matdim.x*matdim.y*sizeof(bool)); // x*y elements in a 1D array
	if (status != hipSuccess){printf("Wall hipMalloc: %s \n", hipGetErrorString(status));}
	// copy wall locations
	status = hipMemcpy(dev_wall, host_Wall, matdim.x*matdim.y*sizeof(bool), hipMemcpyHostToDevice);
	if (status != hipSuccess){printf("Wall hipMemcpy: %s \n", hipGetErrorString(status));}

	// Allocate and initialize arrays for WWall and W
	float * dev_WWall;
	float * dev_W;
	status = hipMalloc((void**)&dev_WWall, WWAL_DIMx*WWAL_DIMy*sizeof(float));
	if (status != hipSuccess){printf("WWall hipMalloc: %s \n", hipGetErrorString(status));}
	status = hipMalloc((void**)&dev_W, W_DIMx*W_DIMy*sizeof(float));
	if (status != hipSuccess){printf("W hipMalloc: %s \n", hipGetErrorString(status));}

	status = hipMemcpy(dev_WWall, host_WWall, WWAL_DIMx*WWAL_DIMy*sizeof(float), hipMemcpyHostToDevice);
	if (status != hipSuccess){printf("WWall hipMemcpy: %s \n", hipGetErrorString(status));}
	status = hipMemcpy(dev_W, host_W, W_DIMx*W_DIMy*sizeof(float), hipMemcpyHostToDevice);
	if (status != hipSuccess){printf("W hipMemcpy: %s \n", hipGetErrorString(status));}
	double source = 0;
	for (int iter = 0; iter < gpu_iterations; iter++)
	{
		printf("Iteration %d: \n", iter);
		PF_iteration_kernel<<<grids,threads>>>(m_device, m_extent, matdim, source, src_loc, dev_wall, dev_WWall, dev_W, nm_device);
		/*	__global__ void PF_iteration_kernel(hipPitchedPtr mPtr, hipExtent mExt, dim3 matrix_dimensions, 
									double src, dim3 srcloc, float * wallLoc, float * WWall, float * W,
									hipPitchedPtr nmPtr)*/
		//PF_iteration_kernel<<<grids,threads>>>(iter, m_device, m_extent, matdim, source);
		source = src_amplitude * sin(2 * PI * src_frequency * (double)(iter) * 0.01);
		printf("Source: %f \n", source);
		hipDeviceSynchronize(); // For getting calculation results back
	}	

	// copy back
	// hipMemcpy(m_host, m_device.ptr, MATRIX_DIM*MATRIX_DIM*4*sizeof(float), hipMemcpyDeviceToHost);

	// Free all allocated memory (move into separate delete function later)
	hipFree(m_device.ptr);
	hipFree(nm_device.ptr);
	hipFree(dev_wall);
	hipFree(dev_WWall);
	hipFree(dev_W);
}

void cPFinit(double matrixFlow[][4], double matrixWall[][4], double in_sourceLoc[])
{
	// Initialize some values
	coef = 1;
	src_amplitude = 1.0;
	src_frequency = 1.0;

	host_Wall = (bool *)malloc(sizeof(bool)*MATRIX_DIM*MATRIX_DIM); 
	memset(host_Wall, 0, MATRIX_DIM*MATRIX_DIM*sizeof(bool));

	host_WWall = (float *)malloc(sizeof(float)*WWAL_DIMx*WWAL_DIMy);
	host_W = (float *)malloc(sizeof(float)*W_DIMx*W_DIMy);
	
	for (int y = 0; y < WWAL_DIMy; y++)
	{
		for (int x = 0; x < WWAL_DIMx; x++)
		{
			host_WWall[x+y*WWAL_DIMx] = matrixWall[x][y]* (coef/2.0);
			host_W[x+y*W_DIMx] = matrixFlow[x][y]* (coef/2.0);
		}
	}

	// copy source loc:
	src_loc.x = in_sourceLoc[0];
	src_loc.y = in_sourceLoc[1];
}

void cPFaddWallLocation(int x, int y, bool val)
{
	if (host_Wall != NULL)	host_Wall[x+y*MATRIX_DIM] = val;
}

void cPFdelete(void)
{
	/*if (host_W != NULL) */free(host_W);
	/*if (host_WWall != NULL) */free(host_WWall);
	/*if (host_Wall != NULL) */free(host_Wall);
	free(m_host);
}

__global__ void testKernel(int a, int b, int *c)
{
	*c = a+b;
	int i = threadIdx.x;
	printf("hello from thread %d \n", i);
}

void callerblahblah(void)
{
	int c;
	int *dev_c;
	int dev;
	//HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
	hipMalloc((void**)&dev_c, sizeof(int));
	testKernel<<<1,1>>>(2,7,dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2+7 = %d \n",c);
	hipFree(dev_c);

	//PF_iteration_kernel<<<1,1>>>();
}